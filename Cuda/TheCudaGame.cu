#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<fstream>

#include "kernel.h"
#include "kernel.cu"

using namespace std;

int main(void) {

    ifstream file;
    file.open("output.txt");
    if (!file.is_open()) {
        cerr << "Could not read file" << endl;
        return -1;
    }

    char separator;
    char playerChar;
    file >> playerChar;
    int player = (playerChar == 'W' ? WHITE : BLACK);
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 

    int xDim, yDim;
    file >> xDim;
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 
    file >> yDim;
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 

    // Supposing there is at least one field
    vector<int> fieldList(xDim * yDim);
    int fieldCount = 0;

    char pos;
    file >> pos;
    if (pos == FIELD_SEPARATOR)
        fieldCount++;
    else if (pos == 'W')
        fieldList.push_back(WHITE);
    else if (pos == 'S')
        fieldList.push_back(BLACK);
    else if (pos == '0')
        fieldList.push_back(WHITE);
    else {
        cerr << "File has wrong format" << endl;
        return(-1);
    }    

    file.close();
    printf("Field count: %d\n", fieldCount);
    cout << "Field count: " << fieldCount << endl;
    cout << "XDim: " << xDim << ", YDim: " << yDim << endl;
    cout << "Starting player: " << player << endl;

    // Run simulation on device
    //vector<bool> results = gameRunner(fieldList, fieldCount, xDim, yDim, player);
}