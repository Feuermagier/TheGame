#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<fstream>

#include "kernel.h"
#include "kernel.cu"

using namespace std;

int main(void) {

    ifstream file;
    file.open("output.txt");
    if (!file.is_open()) {
        cerr << "Could not read file" << endl;
        return -1;
    }

    char separator;
    char playerChar;
    file >> playerChar;
    int player = (playerChar == 'W' ? WHITE : BLACK);
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 

    int xDim, yDim;
    file >> xDim;
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 
    file >> yDim;
    file >> separator;
    if (separator != INIT_SEPARATOR) {
        cerr << "File has an invalid format. Continuing parsing." << endl;
    } 

    vector<int> fieldList(0);
    int fieldCount = 0;

    char pos;
    while(file >> pos) {
        if (pos == FIELD_SEPARATOR)
            fieldCount++;
        else if (pos == 'W')
            fieldList.push_back(WHITE);
        else if (pos == 'S')
            fieldList.push_back(BLACK);
        else if (pos == '0')
            fieldList.push_back(EMPTY);
        else {
            cerr << "File has wrong format" << endl;
            return(-1);
        }    
    }

    file.close();
    cout << "Field count: " << fieldCount << endl;
    cout << "XDim: " << xDim << ", YDim: " << yDim << endl;
    cout << "Starting player: " << player << endl;

    // Run simulation on device
    vector<int> results = gameRunner(fieldList, fieldCount, xDim, yDim, player);

    
    for (int i = 0; i < results.size(); i++) {
        cout << i << ": " << results[i] << endl;
    }

    return 0;
}