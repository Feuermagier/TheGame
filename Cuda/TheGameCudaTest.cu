#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <math.h>
#include <chrono>

const int WHITE = 1;
const int BLACK = -1;
const int EMPTY = 0;

const int LEFT = -1;
const int STRAIGHT = 0;
const int RIGHT = 1;

//int turnCount = 0;
//int gameCount = 0;

///////////////////////// Helpers ////////////////////////////////////

__device__ __host__
void writeValueToFieldList(int* list, int index, int x, int y, int xDim, int yDim, int newValue) {
    *(list + xDim * yDim * index + x * yDim + y) = newValue;
}

__device__ __host__
int* getFieldFromList(int* list, int index, int xDim, int yDim) {
    return list + xDim * yDim * index;
}


__device__ __host__
int getValueAt(int x, int y, int* field, int xDim, int yDim) {
    return field[x * yDim + y];
}

__device__ __host__
void setValueAt(int x, int y, int newValue, int* field, int xDim, int yDim) {
    field[x * yDim + y]; = newValue;
}

__device__ __host__
void copyArrayTo(int* field, int* loc, int xDim, int yDim) {
    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            setValueAt(x, y, getValueAt(x, y, field, xDim, yDim), loc, xDim, yDim);
        }
    }
}

__device__ __host__
int* addFieldToList(int* list, int listSize, int* field, int xDim, int yDim) {
    //int* newList = (int*) realloc(list, (listSize + 1) * xDim * yDim * sizeof(int));
    int* newList = (int*) malloc((listSize + 1) * xDim * yDim * sizeof(int));
    memcpy(newList, list, listSize * xDim * yDim * sizeof(int));
    free(list);
    int* tmp = getFieldFromList(list, listSize, xDim, yDim);
    copyArrayTo(field, tmp, xDim, yDim);
    return newList;
}

__device__ __host__
int* arrayCopy(int* array, int xDim, int yDim) {
    int* copied = (int*) malloc(xDim * yDim * sizeof(int));
    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            setValueAt(x, y, getValueAt(x, y, array, xDim, yDim), copied, xDim, yDim);
        }
    }
    return copied;
}

__device__ __host__
void printTabs(int count) {
    for (int i = 0; i < count ; i++) {
        printf("|  ");
    }
}

__device__ __host__
void printField(int* field, int xDim, int yDim, int tabs) {
    for (int x = 0; x < xDim; x++) {
        printTabs(tabs);
        printf("|");
        for (int y = 0; y < yDim; y++) {
            int position = getValueAt(x, y, field, xDim, yDim);
            printf("  % d", position);
        }
        printf("  |\n");
    }
}


bool fieldEquals(int* fieldOne, int* fieldTwo, int xDim, int yDim) {
    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            if (getValueAt(x, y,fieldOne, xDim, yDim) != getValueAt(x, y, fieldTwo, xDim, yDim))
                return false;
        }
    }
    return true;
}


//////////////////////////////////////////////////////////////////////////////

// Returns true if the given player can win
__device__
bool executeTurn(int* field, int xDim, int yDim, int player, int depth) {
    /*
    if (depth <= 3) {
        printTabs(depth);
        printf("%d's turn:\n", player);
        printField(field, xDim, yDim, depth);
    }
    */
    printf("Hey 1\n");
    int moveCount = 0;
    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            if (getValueAt(x, y, field, xDim, yDim) == player) {
                printf("Hey 2\n");
                // Try all movements
                for (int dir = -1; dir <= 1; dir++) {
                    int xNew = x - player;
                    int yNew = y + dir;

                    // Check if the new position is inside the field boundaries
                    if (xNew >= 0 && xNew < xDim && yNew >= 0 && yNew < yDim) {

                        // Store state of the target position
                        int newPosition = getValueAt(xNew, yNew, field, xDim, yDim);

                        printf("Hey 3\n");

                        // Check if the turn is legal
                        int moveAllowed = abs(dir) + (newPosition * player);  // Zero if you move straight and the target position is empty 
                        // or you move to the right/left and the target position is occupied by an enemy

                        // True if moveAllowed == 0
                        if (!moveAllowed) {
                            //moveCount++;

                            //printTabs(depth);
                            //printf("%d moving from %d,%d to %d,%d\n\n", player, x, y, xNew, yNew);

                            // Set new positions
                            setValueAt(x, y, EMPTY, field, xDim, yDim);
                            setValueAt(xNew, yNew, player, field, xDim, yDim);

                            printf("Hey 4\n");

                            // Check if a win condition is reached
                            if (xNew == (1 - player)/2 * (xDim - 1)) {
                                // Revert changes
                                setValueAt(x, y, player, field, xDim, yDim);
                                setValueAt(xNew, yNew, newPosition, field, xDim, yDim);

                                printf("Hey 5\n");

                                return true;
                            }

                            // Check if the enemy cannot win after this turn, then return true: If you execute this turn, you will win
                            bool canEnemyWin = executeTurn(field, xDim, yDim, -player, depth + 1);
                            // Revert changes
                            setValueAt(x, y, player, field, xDim, yDim);
                            setValueAt(xNew, yNew, newPosition, field, xDim, yDim);

                            printf("Hey 6\n");

                            if (!canEnemyWin) {
                                //printTabs(depth);
                                //printf("--> %d wins\n\n", player);
                                return true;
                            }
                        }
                    }
                }
            }
        }
    }
    //if (moveCount == 0) {
        //gameCount++;
        //turnCount += depth;
    //}
    //printf("\n\n");
    printf("Hey 7\n");
    return false;
}


__global__
void kernel(int* fields, int fieldCount, int xDim, int yDim, int player, bool* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < fieldCount; i += stride) {
        *(result + i) = false;
        bool canWin = executeTurn(getFieldFromList(fields, i, xDim, yDim), xDim, yDim, player, 0);
        if (canWin) {
            *(result + i) = true;
            printf("Wins: %d\n", i);
        } else {
            *(result + i) = false;
        }
    }
}

int main(void) {
    /*
    int xDim = 3, yDim = 3;
    int* field = (int*) malloc(xDim * yDim * sizeof(int));
    memset(field, 0, xDim * yDim * sizeof(int));        // Init array with zeros



    // Init player positions -> one row per player
    for (int i = 0; i < yDim; i++) {
        setValueAt(0, i, BLACK, field, xDim, yDim);
        //setValueAt(1, i, BLACK, field, xDim, yDim);
        //setValueAt(xDim - 2, i, WHITE, field, xDim, yDim);
        setValueAt(xDim - 1, i, WHITE, field, xDim, yDim);
    }

    printField(field, xDim, yDim, 0);
    printf("\n\n");
    */

    /*
    int player = WHITE;

    auto startTime = std::chrono::high_resolution_clock::now();
    bool result = executeTurn(field, xDim, yDim, WHITE, 1);
    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();

    // Outputs
    char* playerText = (player == WHITE) ? "White" : "Black";
    char* resultText = result ? " wins" : " loses";

    std::cout << std::endl << std::endl;
    std::cout << playerText << resultText << std::endl << std::endl;

    std::cout << "Metrics:" << std::endl;
    std::cout << " Elapsed time: " << duration << " microseconds" << std::endl;
    std::cout << " Game count: " << gameCount << std::endl;
    std::cout << " Turn count: " << turnCount << std::endl;

    // Delete field
    free(field);
    */


/*

    int turnCount = 0;
    int player = WHITE;
    int* turnList = determineTurns(field, xDim, yDim, player, &turnCount);

    for (int i = 0; i < 1; i++) {
        int* newList = (int*) malloc(1);
        int newListSize = 0;
        for (int j = 0; j < turnCount; j++) {
            std::cout << "Iteration" << std::endl;
            int count;
            int* tempField = getFieldFromList(turnList, j, xDim, yDim);
            std::cout << "Field created" << std::endl;
            printField(tempField, xDim, yDim, 1);
            int* turns = determineTurns(tempField, xDim, yDim, player, &count);
        
            std::cout << "I'm here!" << std::endl;
            int* tempList = (int*) malloc(1);
            int tempListSize = 0;
            for (int k = 0; k < newListSize + count; k++) {
                if (k < newListSize) {
                    addFieldToList(tempList, k, getFieldFromList(newList, k, xDim, yDim), xDim, yDim);
                } else {
                    addFieldToList(tempList, k, getFieldFromList(turns, k, xDim, yDim), xDim, yDim);
                }
                tempListSize++;
            }
            newList = tempList;
            newListSize = tempListSize;
            //free(tempList);
            std::cout << "I'm here! 2" << std::endl;
        }
        turnList = newList;
        free(newList);
        turnCount = newListSize;
        player = -player;
        std::cout << "I'm here! 3" << std::endl;
    }

    std::cout << "Turn count: " << turnCount << std::endl;
    for (int i = 0; i < turnCount; i++) {
        std::cout << std::endl;
        printField(getFieldFromList(turnList, i, xDim, yDim), xDim, yDim, 0);
    }
    */


    std::ifstream file("output.txt");
    if (!file.is_open()) {
        return -1;
    }
    int xDim = file.get() - '0';
    file.get();
    int yDim = file.get() - '0';
    //file.get();

    int* turnList = (int*) malloc(1);
    int totalFieldCount = 0;

    char pos;
    int x = 0, y = 0;
    int* field = turnList;
    while (file.get(pos)) {
        if (pos == ';') {
            turnList = (int*) realloc(turnList, (totalFieldCount + 1) * xDim * yDim * sizeof(int));
            x = 0;
            y = 0;
            field = turnList + totalFieldCount * xDim * yDim;
            totalFieldCount++;
        } else { 
            if (pos == '0') {
                setValueAt(x, y, EMPTY, field, xDim, yDim);
            } else if (pos == 'S') {
                setValueAt(x, y, BLACK, field, xDim, yDim);
            } else if (pos == 'W') {
                setValueAt(x, y, WHITE, field, xDim, yDim);
            }
            y++;
            if (y >= yDim) {
                x++;
                y = 0;
            }
            //printField(field, xDim, yDim, 0);
        }
    }
    file.close();
    // Supposing there's no ';' at the end of the file
    printf("Field count: %d\n", totalFieldCount);

    /*
    for (int i = 0; i < totalFieldCount; i++) {
        std::cout << std::endl;
        printField(getFieldFromList(turnList, i, xDim, yDim), xDim, yDim, 0);
    }
    */

    int* deviceFieldList;
    hipMallocManaged(&deviceFieldList, totalFieldCount * xDim * yDim * sizeof(int));
    hipMemcpy(deviceFieldList, turnList, totalFieldCount * xDim * yDim * sizeof(int), hipMemcpyHostToDevice);

    /*
    for (int i = 0; i < totalFieldCount; i++) {
        std::cout << std::endl;
        printField(getFieldFromList(deviceFieldList, i, xDim, yDim), xDim, yDim, 0);
    }
    */

    bool* result;
    hipMallocManaged(&result, totalFieldCount * sizeof(bool));
    hipMemset(result, false, totalFieldCount * sizeof(bool));

    int blockSize = 256;
    int numBlocks = (totalFieldCount + blockSize - 1) / blockSize;
    kernel<<<numBlocks, blockSize>>>(deviceFieldList, totalFieldCount, xDim, yDim, WHITE, result);

    hipDeviceSynchronize();

    for (int i = 0; i < totalFieldCount; i++) {
        int r = *(result + i);
        printf("%d: %s\n", i, r ? "true" : "false");
    }
    //printf("%d: %s\n", i, *result ? "true" : "false");

    hipFree(deviceFieldList);
    hipFree(result);
    free(turnList);
}































int* determineTurns(int* field, int xDim, int yDim, int player, int* count) {

    printf("Before list init\n");
    int* list = (int*) malloc(1);
    printf("After after init\n");

    int i = 0;
    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            if (getValueAt(x, y, field, xDim, yDim) == player) {
                // Try all movements
                for (int dir = -1; dir <= 1; dir++) {
                    int xNew = x - player;
                    int yNew = y + dir;

                    // Check if the new position is inside the field boundaries
                    if (xNew >= 0 && xNew < xDim && yNew >= 0 && yNew < yDim) {

                        // Store state of the target position
                        int newPosition = getValueAt(xNew, yNew, field, xDim, yDim);

                        // Check if the turn is legal
                        int moveAllowed = abs(dir) + (newPosition * player);  // Zero if you move straight and the target position is empty 
                        // or you move to the right/left and the target position is occupied by an enemy

                        // True if moveAllowed == 0
                        if (!moveAllowed) {
                            printf("Moving\n");
                            // Set new positions
                            setValueAt(x, y, EMPTY, field, xDim, yDim);
                            setValueAt(xNew, yNew, player, field, xDim, yDim);

                            printf("Positions set\n");
                            list = (int*) realloc(list, (i+1) * xDim * yDim * sizeof(int));
                            printf("Memory allocated\n");
                            copyArrayTo(field, getFieldFromList(list, i, xDim, yDim), xDim, yDim);
                            i++;
                            printf("Array copied\n");

                            // Revert changes
                            setValueAt(x, y, player, field, xDim, yDim);
                            setValueAt(xNew, yNew, newPosition, field, xDim, yDim);
                            printf("Reverted changes\n\n");
                        }
                    }
                }
            }
        }
    }
    *count = i;
    return list;
}
