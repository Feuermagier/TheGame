#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include <vector>

#include "hip/hip_runtime.h"
#include "kernel.h"
#include "dev_array.h"
#include "constants.h"

using namespace std;


// Expensive, use not that often
// Returns a pointer to a newly allocated array for the field
__device__
int* getFieldFromList(int* list, int index, int xDim, int yDim) {
    int* field = (int*) malloc(xDim * yDim * sizeof(int));
    memcpy(field, &list[xDim * yDim * index], xDim * yDim * sizeof(int));
    return field;
}


__device__
int getValueAt(int x, int y, int* field, int xDim, int yDim) {
    return field[x * yDim + y];
}

__device__ 
void setValueAt(int x, int y, int newValue, int* field, int xDim, int yDim) {
    field[x * yDim + y] = newValue;
}

// Returns true if the given player can win
__device__
bool executeTurn(int* field, int xDim, int yDim, int player, int depth) {

    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            if (getValueAt(x, y, field, xDim, yDim) == player) {

                // Try all movements
                for (int dir = -1; dir <= 1; dir++) {
                    int xNew = x - player;
                    int yNew = y + dir;

                    // Check if the new position is inside the field boundaries
                    if (xNew >= 0 && xNew < xDim && yNew >= 0 && yNew < yDim) {

                        // Store state of the target position
                        int newPosition = getValueAt(xNew, yNew, field, xDim, yDim);


                        // Check if the turn is legal
                        int moveAllowed = abs(dir) + (newPosition * player);  // Zero if you move straight and the target position is empty 
                        // or you move to the right/left and the target position is occupied by an enemy

                        // True if moveAllowed == 0
                        if (!moveAllowed) {

                            // Set new positions
                            setValueAt(x, y, EMPTY, field, xDim, yDim);
                            setValueAt(xNew, yNew, player, field, xDim, yDim);


                            // Check if a win condition is reached
                            if (xNew == (1 - player)/2 * (xDim - 1)) {
                                // Revert changes
                                setValueAt(x, y, player, field, xDim, yDim);
                                setValueAt(xNew, yNew, newPosition, field, xDim, yDim);

                                return true;
                            }

                            // Check if the enemy cannot win after this turn, then return true: If you execute this turn, you will win
                            bool canEnemyWin = executeTurn(field, xDim, yDim, -player, depth + 1);
                            // Revert changes
                            setValueAt(x, y, player, field, xDim, yDim);
                            setValueAt(xNew, yNew, newPosition, field, xDim, yDim);

                            if (!canEnemyWin) {
                                return true;
                            }
                        }
                    }
                }
            }
        }
    }
    return false;
}

///////////////////////////// KERNEL ////////////////////////////////////////
__global__
void gameRunnerKernel(int* fieldList, int fieldCount, int xDim, int yDim, int player, int* results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < fieldCount; i += stride) {
        // Copy field to local memory
        int* field = getFieldFromList(fieldList, i, xDim, yDim);
        // Run game simulation
        bool canWin = executeTurn(field, xDim, yDim, player, 0);
        // Set result array according to the result
        if (canWin) {
            results[i] = 1;
            printf("Wins: %d\n", i);
        } else {
            results[i] = 0;
        }
    }
}

vector<int> gameRunner(vector<int> fieldList, int fieldCount, int xDim, int yDim, int player) {

    // Copy field data to the device
    dev_array<int> devFieldList(fieldList.size());
    devFieldList.set(&fieldList[0], fieldList.size());

    // Create and copy result array
    vector<int> results(fieldCount);
    dev_array<int> deviceResults(fieldCount);
    deviceResults.set(&results[0], fieldCount);


    int blockSize = 256;
    int numBlocks = (fieldCount + blockSize - 1) / blockSize;
    gameRunnerKernel<<<numBlocks, blockSize>>>(devFieldList.getData(), fieldCount, xDim, yDim, player, deviceResults.getData());

    // Copy data back to the host
    deviceResults.set(&results[0], fieldCount);

    hipDeviceSynchronize();

    return results;
}