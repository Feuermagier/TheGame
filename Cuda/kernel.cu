#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include <vector>

#include "hip/hip_runtime.h"
#include "kernel.h"
#include "dev_array.h"
#include "constants.h"

using namespace std;


__device__
int getValueAt(int x, int y, int* field, int xDim, int yDim) {
    return field[x * yDim + y];
}

__device__ 
void setValueAt(int x, int y, int newValue, int* field, int xDim, int yDim) {
    field[x * yDim + y] = newValue;
}

// Returns true if the given player can win
__device__
bool executeTurn(int* field, int xDim, int yDim, int player, int depth) {

    for (int x = 0; x < xDim; x++) {
        for (int y = 0; y < yDim; y++) {
            if (getValueAt(x, y, field, xDim, yDim) == player) {
                // Try all movements
                //printf("Found one at %d, %d\n", x, y);
                for (int dir = -1; dir <= 1; dir++) {
                    int xNew = x - player;
                    int yNew = y + dir;

                    // Check if the new position is inside the field boundaries
                    if (xNew >= 0 && xNew < xDim && yNew >= 0 && yNew < yDim) {
                        // Store state of the target position
                        int newPosition = getValueAt(xNew, yNew, field, xDim, yDim);


                        // Check if the turn is legal
                        int moveAllowed = abs(dir) + (newPosition * player);  // Zero if you move straight and the target position is empty 
                        // or you move to the right/left and the target position is occupied by an enemy

                        // True if moveAllowed == 0
                        if (!moveAllowed) {
                            //printf("Can move from %d, %d to %d, %d\n", x, y, xNew, yNew);

                            // Set new positions
                            setValueAt(x, y, EMPTY, field, xDim, yDim);
                            setValueAt(xNew, yNew, player, field, xDim, yDim);

                            // Check if a win condition is reached
                            if (xNew == (1 - player)/2 * (xDim - 1)) {
                                // Revert changes
                                setValueAt(x, y, player, field, xDim, yDim);
                                setValueAt(xNew, yNew, newPosition, field, xDim, yDim);
                                return true;
                            }

                            // Check if the enemy cannot win after this turn, then return true: If you execute this turn, you will win
                            bool canEnemyWin = executeTurn(field, xDim, yDim, -1 * player, depth + 1);
                            // Revert changes
                            setValueAt(x, y, player, field, xDim, yDim);
                            setValueAt(xNew, yNew, newPosition, field, xDim, yDim);
                            if (!canEnemyWin) {
                                return true;
                            }
                        }
                    }
                }
            }
        }
    }
    return false;
}

///////////////////////////// KERNEL ////////////////////////////////////////
__global__
void gameRunnerKernel(int* fieldList, int fieldCount, int xDim, int yDim, int player, int* results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    extern __shared__ int fieldCache[];
    //int* field = (int*) malloc(xDim * yDim * sizeof(int));

    for (int i = index; i < fieldCount; i += stride) {
        // Copy field to local memory
        memcpy(&fieldCache[threadIdx.x * xDim * yDim], &fieldList[xDim * yDim * i], xDim * yDim * sizeof(int));
        /*
        printf("%d %d\n", field[0], field[1]);
        printf("%d %d\n", field[2], field[3]);
        printf("%d %d\n", field[4], field[5]);
        printf("%d %d\n", field[6], field[7]);
        printf("%d %d\n", field[8], field[9]);
        printf("%d %d\n\n", field[10], field[11]);    
        */

        // Run game simulation
        bool canWin = executeTurn(&fieldCache[threadIdx.x * xDim * yDim], xDim, yDim, player, 0);
        // Set result array according to the result
        if (canWin) {
            results[i] = 1;
            printf("Wins: %d\n", i);
        } else {
            results[i] = 0;
        }
    }
}

vector<int> gameRunner(vector<int> fieldList, int fieldCount, int xDim, int yDim, int player) {

    /*
    // Copy field data to the device
    dev_array<int> devFieldList(fieldList.size());
    devFieldList.set(&fieldList[0], fieldList.size());

    // Create and copy result array
    vector<int> results(fieldCount);
    dev_array<int> deviceResults(fieldCount);
    deviceResults.set(&results[0], fieldCount);
    */

    int* devFieldList;
    hipMallocManaged(&devFieldList, fieldList.size() * sizeof(int));
    hipMemcpy(devFieldList, &fieldList[0], fieldList.size() * sizeof(int), hipMemcpyHostToDevice);

    vector<int> results(fieldCount);
    int* deviceResults;
    hipMallocManaged(&deviceResults, fieldCount * sizeof(int));
    hipMemcpy(deviceResults, &results[0], fieldCount * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int blockCount = (fieldCount + blockSize - 1) / blockSize;
    gameRunnerKernel<<<blockCount, blockSize, blockSize * xDim * yDim * sizeof(int)>>>(devFieldList, fieldCount, xDim, yDim, player, deviceResults);
    //gameRunnerKernel<<<1, 32>>>(devFieldList, fieldCount, xDim, yDim, player, deviceResults);

    error = hipDeviceSynchronize();
    cout << error << endl;

    // Copy data back to the host
    //deviceResults.set(&results[0], fieldCount);
    hipMemcpy(&results[0], deviceResults, fieldCount * sizeof(int), hipMemcpyDeviceToHost);

    return results;
}